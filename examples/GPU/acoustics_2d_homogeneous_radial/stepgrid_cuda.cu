#include "real.H"
#include "params.H"
#include "acoustics_riemann_solver.H"
#include "fused_solvers_headers.H"
#include "advance.H"
#include <hip/hip_runtime.h>

extern "C" void call_C_limited_riemann_update(
        const int cellsX, const int cellsY, const int ghostCells,
        const real startX, const real endX, const real startY, const real endY,
        const real dt,
        real* q, real* qNew, 
        real* coefficients,
        real* waveSpeedsX, real* waveSpeedsY,
        const int numStates, const int numCoefficients,
        real* cfls, const int ngrids, const int mcapa,
        const int id, const int dev_id) {

    // actually qNew holds the input old solution as well as new output solution
    // q is just temporary storage for intermediate solution between x-sweep and y-sweep

    hipStream_t stream;

    get_cuda_stream(id, dev_id, &stream);

    pdeParam param(cellsX, cellsY, ghostCells, 
            numStates, NWAVES, numCoefficients,
            startX, endX, startY, endY, dt,
            q, qNew, 
            coefficients, 
            waveSpeedsX, waveSpeedsY,
            cfls, mcapa, id, dev_id);

    param.setOrderOfAccuracy(2);

    acoustics_homo_2d_horizontal acoustic_h;
    acoustics_homo_2d_vertical acoustic_v;
    
    limiter_VanLeer phi;

    limited_Riemann_Update(param, 
            acoustic_h, acoustic_v, 
            phi,stream);

}

